#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>
#include <glm/mat3x3.hpp>
#include <glm/vec3.hpp>

#include "svd.h"
#include "cuda_error.h"

using namespace glm;

__global__
void test_kernel(mat3 *matrix, vec3 *s, mat3 *v)
{
	printf("%p %f\n", s, (*s)[2]);

	if(threadIdx.x == 0) {
		dsvd<3, 3>(*matrix, *s, *v);
	}
}

void test(mat3 &m, vec3 &s, mat3 &v)
{
    // printf("%f %f %f\n", m[0][0], m[1][1], m[2][2]);
	mat3* dm;
	vec3* ds;
	mat3* dv;
	CudaSafeCall(hipMalloc(&dm, sizeof(mat3)));
	CudaSafeCall(hipMalloc(&ds, sizeof(vec3)));
	CudaSafeCall(hipMalloc(&dv, sizeof(mat3)));
	std::cout << ds << std::endl;
	CudaSafeCall(hipMemcpy(dm, &m, sizeof(mat3), hipMemcpyHostToDevice));
	CudaSafeCall(hipMemset(ds, 0, sizeof(vec3)));
	CudaSafeCall(hipMemset(dv, 0, sizeof(mat3)));
	Launch(test_kernel<<<1, 1>>>(dm, ds, dv));
	CudaSafeCall(hipMemcpy(&m, dm, sizeof(mat3), hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy(&s, ds, sizeof(vec3), hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy(&v, dv, sizeof(mat3), hipMemcpyDeviceToHost));
}